#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>


#include "math.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <iomanip>

int main(void){
    hipDeviceProp_t property;

    int count;
    hipGetDeviceCount(&count);
    printf("Count: "+count);
    for(int i=0;i<count;i++){
        hipGetDeviceProperties(&property, i);
    }
    return 0;
}
